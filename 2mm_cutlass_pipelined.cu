/*****************************************************************************
 * 2mm_cutlass.cu  –  PolyBench “2 MM” solved with CUTLASS tensor‑core GEMM
 *
 *  D = β·D + α·(A·B)·C
 * 
 * Compile with the following flag: -I$CUTLASS_DIR/include
 * $CUTLASS_DIR is the path to the CUTLASS library.
 * 
 * Clone the CUTLASS repository from: https://github.com/NVIDIA/cutlass.git
 * 
 * 
 * Note: The code is based on the CUTLASS library which inherently takes care of pipelining.
 *       Since we're using pre-existing CUTLASS kernels, the code is automatically pipelined.
*****************************************************************************/

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

#define POLYBENCH_TIME 1
#include "2mm.cuh"
#include "../../common/polybench.h"
#include "../../common/polybenchUtilFuncts.h"

#include <cutlass/layout/matrix.h>
#include <cutlass/gemm/device/gemm.h>

using RowMajor = cutlass::layout::RowMajor;

using GemmRRR = cutlass::gemm::device::Gemm<float, RowMajor, float, RowMajor, float, RowMajor, float>; 

#define CUTLASS_CHECK(status)                               \
  {if (status != cutlass::Status::kSuccess) {                \
      printf("CUTLASS error: %d\n", int(status));             \
      std::exit(EXIT_FAILURE); }}

/* --------------------------------------------------------------------- */
/*  GPU version using two CUTLASS GEMMs                                  */
/* --------------------------------------------------------------------- */
void mm2Cuda(int ni,int nj,int nk,int nl, float alpha, float beta,
            float POLYBENCH_2D(tmp,NI,NJ,ni,nj), float POLYBENCH_2D(A,NI,NK,ni,nk),
            float POLYBENCH_2D(B,NK,NJ,nk,nj), float POLYBENCH_2D(C,NL,NJ,nl,nj),
            float POLYBENCH_2D(D,NI,NL,ni,nl), float POLYBENCH_2D(D_gpu,NI,NL,ni,nl))
{
    float *dA,*dB,*dC,*dTmp,*dD;
    hipMalloc(&dA, NI*NK*sizeof(float));
    hipMalloc(&dB, NK*NJ*sizeof(float));
    hipMalloc(&dC, NL*NJ*sizeof(float));
    hipMalloc(&dTmp,NI*NJ*sizeof(float));
    hipMalloc(&dD, NI*NL*sizeof(float));

    hipMemcpy(dA,A, NI*NK*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB,B, NK*NJ*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dC,C, NL*NJ*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dD,D, NI*NL*sizeof(float), hipMemcpyHostToDevice);

    printf("GPU time:\n");
    polybench_start_instruments;

    /*
    Standard format of multiplication in the GemmRRR kernel:
        D (Output) = α·A·B + β·C
        for Gemm1 below: temp = α·A·B + 0·C
        for Gemm2 below: D = 1·temp·C + β·D

        So for Gemm1, (α,β) = (α,0)
        and for Gemm2, (α,β) = (1,β)
    */

    /* GEMM #1 : tmp = α · A · B  ------------------------------------ */
    GemmRRR gemmAB;
    GemmRRR::Arguments argsAB(
        {ni, nj, nk},            // 1) problem shape  (M,N,K)
        {dA,  nk},               // 2) A  (ptr , leading-dim = NK)
        {dB,  nj},               // 3) B  (ptr , leading-dim = NJ)
        {nullptr, 0},            // 4) C  (unused → β·C term is zero)
        {dTmp, nj},              // 5) D  (output buffer, ld = NJ)
        {alpha, 0.0f});          // 6) epilogue scalars (α , 0) ---> α*(A · B) + 0*C    
    CUTLASS_CHECK(gemmAB.initialize(argsAB));
    CUTLASS_CHECK(gemmAB());

    /* GEMM #2 : D = 1·tmp·C + β·D  ---------------------------------- */
    GemmRRR gemm2;
    GemmRRR::Arguments args2(
        {ni, nl, nj},            // (M,N,K)  = (NI,NL,NJ)
        {dTmp, nj},              // A  (tmp, produced above)
        {dC,   nl},              // B
        {dD,   nl},              // C  (original D, will be scaled by β)
        {dD,   nl},              // D  (output written in-place)
        {1.0f, beta});           // epilogue scalars (1,β) ---> 1*(tmp · C) + β*D
    CUTLASS_CHECK(gemm2.initialize(args2));
    CUTLASS_CHECK(gemm2());

    polybench_stop_instruments;
    polybench_print_instruments;

    hipMemcpy(D_gpu, dD, NI*NL*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dA); 
    hipFree(dB); 
    hipFree(dC); 
    hipFree(dTmp); 
    hipFree(dD);
}

/* --------------------------------------------------------------------- */
void init_array(int ni,int nj,int nk,int nl, float *alpha, float *beta, float POLYBENCH_2D(A,NI,NK,ni,nk), float POLYBENCH_2D(B,NK,NJ,nk,nj), float POLYBENCH_2D(C,NL,NJ,nl,nj), float POLYBENCH_2D(D,NI,NL,ni,nl))
{
    int i,j; *alpha=32412.0f; *beta=2123.0f;
    for(i=0;i<ni;i++) 
    for(j=0;j<nk;j++) 
        A[i][j]=((float)i*j)/NI;

    for(i=0;i<nk;i++) 
    for(j=0;j<nj;j++) 
        B[i][j]=((float)i*(j+1))/NJ;

    for(i=0;i<nl;i++) 
    for(j=0;j<nj;j++) 
        C[i][j]=((float)i*(j+3))/NL;

    for(i=0;i<ni;i++) 
    for(j=0;j<nl;j++) 
        D[i][j]=((float)i*(j+2))/NK;
}

#define PERCENT_DIFF_ERROR_THRESHOLD 0.05f
void compareResults(int ni,int nl, float POLYBENCH_2D(D1,NI,NL,ni,nl), float POLYBENCH_2D(D2,NI,NL,ni,nl))
{
    int fail=0;

    for(int i=0;i<ni;i++) 
    for(int j=0;j<nl;j++)
    if(percentDiff(D1[i][j],D2[i][j])>PERCENT_DIFF_ERROR_THRESHOLD) 
        fail++;

    printf("Non‑matching elements >%.2f%% : %d\n",
    PERCENT_DIFF_ERROR_THRESHOLD, fail);
    }

void GPU_argv_init(){
    hipDeviceProp_t p; 
    hipGetDeviceProperties(&p,0);
    printf("Using GPU 0 : %s\n", p.name);  
    hipSetDevice(0);
}

void mm2_cpu(int ni,int nj,int nk,int nl, float alpha,float beta, float POLYBENCH_2D(tmp,NI,NJ,ni,nj),
            float POLYBENCH_2D(A,NI,NK,ni,nk), float POLYBENCH_2D(B,NK,NJ,nk,nj),
            float POLYBENCH_2D(C,NL,NJ,nl,nj), float POLYBENCH_2D(D,NI,NL,ni,nl))
{
    for(int i=0;i<ni;i++)
        for(int j=0;j<nj;j++){
            tmp[i][j]=0;
        for(int k=0;k<nk;k++)
            tmp[i][j]+=alpha*A[i][k]*B[k][j];
        }

    for(int i=0;i<ni;i++)
        for(int j=0;j<nl;j++){
            D[i][j]*=beta;
        for(int k=0;k<nj;k++)
            D[i][j]+=tmp[i][k]*C[k][j];
    }
}

/* --------------------------------------------------------------------- */
int main()
{
    int ni=NI,nj=NJ,nk=NK,nl=NL;
    float alpha,beta;

    POLYBENCH_2D_ARRAY_DECL(tmp,float,NI,NJ,ni,nj);
    POLYBENCH_2D_ARRAY_DECL(A,float,NI,NK,ni,nk);
    POLYBENCH_2D_ARRAY_DECL(B,float,NK,NJ,nk,nj);
    POLYBENCH_2D_ARRAY_DECL(C,float,NL,NJ,nl,nj);
    POLYBENCH_2D_ARRAY_DECL(D,float,NI,NL,ni,nl);
    POLYBENCH_2D_ARRAY_DECL(D_gpu,float,NI,NL,ni,nl);

    init_array(ni,nj,nk,nl,&alpha,&beta,
            POLYBENCH_ARRAY(A),POLYBENCH_ARRAY(B),
            POLYBENCH_ARRAY(C),POLYBENCH_ARRAY(D));

    GPU_argv_init();

    mm2Cuda(ni,nj,nk,nl,alpha,beta,
            POLYBENCH_ARRAY(tmp),POLYBENCH_ARRAY(A),POLYBENCH_ARRAY(B),
            POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(D), POLYBENCH_ARRAY(D_gpu));

    polybench_start_instruments;
    mm2_cpu(ni,nj,nk,nl,alpha,beta,
            POLYBENCH_ARRAY(tmp),POLYBENCH_ARRAY(A),POLYBENCH_ARRAY(B),
            POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(D));
    printf("CPU time:\n");
    polybench_stop_instruments; 
    polybench_print_instruments;

    compareResults(ni,nl, POLYBENCH_ARRAY(D), POLYBENCH_ARRAY(D_gpu));

    return 0;
}

#include "../../common/polybench.c"
