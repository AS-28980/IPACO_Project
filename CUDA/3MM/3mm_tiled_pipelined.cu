#include "hip/hip_runtime.h"
/**
 * 3mm.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define POLYBENCH_TIME 1
#define GPU_DEVICE 0
#define TILE 32

#include "3mm.cuh"
#include "../../common/polybench.h"
#include "../../common/polybenchUtilFuncts.h"

#define GPU_DEVICE 0

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define RUN_ON_CPU


void init_array(int ni, int nj, int nk, int nl, int nm, DATA_TYPE POLYBENCH_2D(A, NI, NK, ni, nk), DATA_TYPE POLYBENCH_2D(B, NK, NJ, nk, nj), 
        DATA_TYPE POLYBENCH_2D(C, NJ, NM, nj, nm), DATA_TYPE POLYBENCH_2D(D, NM, NL, nm, nl))
{
    int i, j;

    for (i = 0; i < ni; i++)
    {
        for (j = 0; j < nk; j++)
        {
            A[i][j] = ((DATA_TYPE) i*j) / ni;
        }
    }

    for (i = 0; i < nk; i++)
    {
        for (j = 0; j < nj; j++)
        {
            B[i][j] = ((DATA_TYPE) i*(j+1)) / nj;
        }
    }

    for (i = 0; i < nj; i++)
    {
        for (j = 0; j < nm; j++)
        {
            C[i][j] = ((DATA_TYPE) i*(j+3)) / nl;
        }
    }

    for (i = 0; i < nm; i++)
    {
        for (j = 0; j < nl; j++)
        {
            D[i][j] = ((DATA_TYPE) i*(j+2)) / nk;
        }
    }
}


void compareResults(int ni, int nl, DATA_TYPE POLYBENCH_2D(G, NI, NL, ni, nl), DATA_TYPE POLYBENCH_2D(G_outputFromGpu, NI, NL, ni, nl))
{
    int i,j,fail;
    fail = 0;

    for (i=0; i < ni; i++)
    {
        for (j=0; j < nl; j++)
        {
            if (percentDiff(G[i][j], G_outputFromGpu[i][j]) > PERCENT_DIFF_ERROR_THRESHOLD)
            {
                fail++;				
            }
        }
    }
    
    // print results
    printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
    printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
    hipSetDevice( GPU_DEVICE );
}

__global__ void mm3_kernel1_tiled_pipelined(int ni,int nj,int nk, const DATA_TYPE* __restrict__ A, const DATA_TYPE* __restrict__ B, DATA_TYPE* __restrict__ E)
{
    __shared__ DATA_TYPE shA1[TILE][TILE], shA2[TILE][TILE];
    __shared__ DATA_TYPE shB1[TILE][TILE], shB2[TILE][TILE];

    int col = blockIdx.x * TILE + threadIdx.x;

    DATA_TYPE acc1 = 0, acc2 = 0;

    DATA_TYPE *shAptr = &shA1[0][0];
    DATA_TYPE *shBptr = &shB1[0][0];

    if (threadIdx.y < TILE/2) {
        int isub1 = 2 * threadIdx.y;
        int isub2 = isub1 + 1;
        int row1 = blockIdx.y * blockDim.y + isub1;
        int row2 = blockIdx.y * blockDim.y + isub2;

        if (row1 < ni && threadIdx.x < nk) {
            shAptr[isub1*TILE + threadIdx.x] = A[row1 * nk + threadIdx.x];
            shAptr[isub2*TILE + threadIdx.x] = A[row2 * nk + threadIdx.x];
        } else {
            shAptr[isub1*TILE + threadIdx.x] = 0;
            shAptr[isub2*TILE + threadIdx.x] = 0;
        }

        if (col < nj && isub1 < nk) {
            shBptr[isub1*TILE + threadIdx.x] = B[isub1 * nj + col];
            shBptr[isub2*TILE + threadIdx.x] = B[isub2 * nj + col];
        } else {
            shBptr[isub1*TILE + threadIdx.x] = 0;
            shBptr[isub2*TILE + threadIdx.x] = 0;
        }
    }

    __syncthreads();

    DATA_TYPE *shAptr2 = shAptr;
    DATA_TYPE *shBptr2 = shBptr;
    shAptr = &shA2[0][0];
    shBptr = &shB2[0][0];

    for (int t = 0; t < (nk + TILE - 1) / TILE - 1; t++) {
        if (threadIdx.y < TILE/2) {
            int isub1 = 2 * threadIdx.y;
            int isub2 = isub1 + 1;
            int row1 = blockIdx.y * blockDim.y + isub1;
            int row2 = blockIdx.y * blockDim.y + isub2;
            int k = (t + 1) * TILE + threadIdx.x;

            shAptr[isub1*TILE + threadIdx.x] = (row1 < ni && k < nk) ? A[row1 * nk + k] : 0;
            shAptr[isub2*TILE + threadIdx.x] = (row2 < ni && k < nk) ? A[row2 * nk + k] : 0;

            int b_row1 = (t + 1) * TILE + isub1;
            int b_row2 = b_row1 + 1;

            shBptr[isub1*TILE + threadIdx.x] = (col < nj && b_row1 < nk) ? B[b_row1 * nj + col] : 0;
            shBptr[isub2*TILE + threadIdx.x] = (col < nj && b_row2 < nk) ? B[b_row2 * nj + col] : 0;
        } else {
            int isub1 = 2 * (threadIdx.y - TILE/2);
            int isub2 = isub1 + 1;
            for (int k = 0; k < TILE; k++) {
                acc1 += shAptr2[isub1*TILE + k] * shBptr2[k*TILE + threadIdx.x];
                acc2 += shAptr2[isub2*TILE + k] * shBptr2[k*TILE + threadIdx.x];
            }
        }
        DATA_TYPE* tmp;

        tmp = shAptr; shAptr = shAptr2; shAptr2 = tmp;
        tmp = shBptr; shBptr = shBptr2; shBptr2 = tmp;
        __syncthreads();
    }

    if (threadIdx.y >= TILE/2) {
        int isub1 = 2 * (threadIdx.y - TILE/2);
        int isub2 = isub1 + 1;
        for (int k = 0; k < TILE; k++) {
            acc1 += shAptr2[isub1*TILE + k] * shBptr2[k*TILE + threadIdx.x];
            acc2 += shAptr2[isub2*TILE + k] * shBptr2[k*TILE + threadIdx.x];
        }

        int row1 = blockIdx.y * blockDim.y + isub1;
        int row2 = blockIdx.y * blockDim.y + isub2;
        if (row1 < ni && col < nj) E[row1 * nj + col] = acc1;
        if (row2 < ni && col < nj) E[row2 * nj + col] = acc2;
    }
}

__global__ void mm3_kernel2_tiled_pipelined(int nj,int nl,int nm, const DATA_TYPE* __restrict__ C, const DATA_TYPE* __restrict__ D, DATA_TYPE* __restrict__ F)
{
    __shared__ DATA_TYPE shC1[TILE][TILE], shC2[TILE][TILE];
    __shared__ DATA_TYPE shD1[TILE][TILE], shD2[TILE][TILE];

    int col = blockIdx.x * TILE + threadIdx.x;

    DATA_TYPE acc1 = 0, acc2 = 0;

    DATA_TYPE *shCptr = &shC1[0][0];
    DATA_TYPE *shDptr = &shD1[0][0];

    if (threadIdx.y < TILE/2) {
        int isub1 = 2 * threadIdx.y;
        int isub2 = isub1 + 1;
        int row1 = blockIdx.y * blockDim.y + isub1;
        int row2 = blockIdx.y * blockDim.y + isub2;

        if (row1 < nj && threadIdx.x < nm) {
            shCptr[isub1*TILE + threadIdx.x] = C[row1 * nm + threadIdx.x];
            shCptr[isub2*TILE + threadIdx.x] = C[row2 * nm + threadIdx.x];
        } else {
            shCptr[isub1*TILE + threadIdx.x] = shCptr[isub2*TILE + threadIdx.x] = 0;
        }

        if (col < nl && isub1 < nm) {
            shDptr[isub1*TILE + threadIdx.x] = D[isub1 * nl + col];
            shDptr[isub2*TILE + threadIdx.x] = D[isub2 * nl + col];
        } else {
            shDptr[isub1*TILE + threadIdx.x] = shDptr[isub2*TILE + threadIdx.x] = 0;
        }
    }

    __syncthreads();

    DATA_TYPE *shCptr2 = shCptr;
    DATA_TYPE *shDptr2 = shDptr;
    shCptr = &shC2[0][0];
    shDptr = &shD2[0][0];


    for (int t = 0; t < (nm + TILE - 1) / TILE - 1; t++) {
        if (threadIdx.y < TILE/2) {
            int isub1 = 2 * threadIdx.y;
            int isub2 = isub1 + 1;
            int row1 = blockIdx.y * blockDim.y + isub1;
            int row2 = blockIdx.y * blockDim.y + isub2;
            int k = (t + 1) * TILE + threadIdx.x;

            shCptr[isub1*TILE + threadIdx.x] = (row1 < nj && k < nm) ? C[row1 * nm + k] : 0;
            shCptr[isub2*TILE + threadIdx.x] = (row2 < nj && k < nm) ? C[row2 * nm + k] : 0;

            int d_row1 = (t + 1) * TILE + isub1;
            int d_row2 = d_row1 + 1;

            shDptr[isub1*TILE + threadIdx.x] = (col < nl && d_row1 < nm) ? D[d_row1 * nl + col] : 0;
            shDptr[isub2*TILE + threadIdx.x] = (col < nl && d_row2 < nm) ? D[d_row2 * nl + col] : 0;
        } else {
            int isub1 = 2 * (threadIdx.y - TILE/2);
            int isub2 = isub1 + 1;
            for (int k = 0; k < TILE; k++) {
                acc1 += shCptr2[isub1*TILE + k] * shDptr2[k*TILE + threadIdx.x];
                acc2 += shCptr2[isub2*TILE + k] * shDptr2[k*TILE + threadIdx.x];
            }
        }

        DATA_TYPE* tmp;

        tmp = shCptr; shCptr = shCptr2; shCptr2 = tmp;
        tmp = shDptr; shDptr = shDptr2; shDptr2 = tmp;
        __syncthreads();
    }

    if (threadIdx.y >= TILE/2) {
        int isub1 = 2 * (threadIdx.y - TILE/2);
        int isub2 = isub1 + 1;
        for (int k = 0; k < TILE; k++) {
            acc1 += shCptr[isub1*TILE + k] * shDptr[k*TILE + threadIdx.x];
            acc2 += shCptr[isub2*TILE + k] * shDptr[k*TILE + threadIdx.x];
        }

        int row1 = blockIdx.y * blockDim.y + isub1;
        int row2 = blockIdx.y * blockDim.y + isub2;
        if (row1 < nj && col < nl) F[row1 * nl + col] = acc1;
        if (row2 < nj && col < nl) F[row2 * nl + col] = acc2;
    }
}

__global__ void mm3_kernel3_tiled_pipelined(int ni,int nj,int nl, const DATA_TYPE* __restrict__ E, const DATA_TYPE* __restrict__ F, DATA_TYPE* __restrict__ G)
{
    __shared__ DATA_TYPE shE1[TILE][TILE], shE2[TILE][TILE];
    __shared__ DATA_TYPE shF1[TILE][TILE], shF2[TILE][TILE];

    int col = blockIdx.x * TILE + threadIdx.x;

    DATA_TYPE acc1 = 0, acc2 = 0;

    DATA_TYPE *shEptr = &shE1[0][0];
    DATA_TYPE *shFptr = &shF1[0][0];

    if (threadIdx.y < TILE/2) {
        int isub1 = 2 * threadIdx.y;
        int isub2 = isub1 + 1;
        int row1 = blockIdx.y * blockDim.y + isub1;
        int row2 = blockIdx.y * blockDim.y + isub2;

        if (row1 < ni && threadIdx.x < nj) {
            shEptr[isub1*TILE + threadIdx.x] = E[row1 * nj + threadIdx.x];
            shEptr[isub2*TILE + threadIdx.x] = E[row2 * nj + threadIdx.x];
        } else {
            shEptr[isub1*TILE + threadIdx.x] = shEptr[isub2*TILE + threadIdx.x] = 0;
        }

        if (col < nl && isub1 < nj) {
            shFptr[isub1*TILE + threadIdx.x] = F[isub1 * nl + col];
            shFptr[isub2*TILE + threadIdx.x] = F[isub2 * nl + col];
        } else {
            shFptr[isub1*TILE + threadIdx.x] = shFptr[isub2*TILE + threadIdx.x] = 0;
        }
    }

    __syncthreads();

    DATA_TYPE *shEptr2 = shEptr;
    DATA_TYPE *shFptr2 = shFptr;
    shEptr = &shE2[0][0];
    shFptr = &shF2[0][0];

    for (int t = 0; t < (nj + TILE - 1) / TILE - 1; t++) {
        if (threadIdx.y < TILE/2) {
            int isub1 = 2 * threadIdx.y;
            int isub2 = isub1 + 1;
            int row1 = blockIdx.y * blockDim.y + isub1;
            int row2 = blockIdx.y * blockDim.y + isub2;
            int k = (t + 1) * TILE + threadIdx.x;

            shEptr[isub1*TILE + threadIdx.x] = (row1 < ni && k < nj) ? E[row1 * nj + k] : 0;
            shEptr[isub2*TILE + threadIdx.x] = (row2 < ni && k < nj) ? E[row2 * nj + k] : 0;

            int f_row1 = (t + 1) * TILE + isub1;
            int f_row2 = f_row1 + 1;

            shFptr[isub1*TILE + threadIdx.x] = (col < nl && f_row1 < nj) ? F[f_row1 * nl + col] : 0;
            shFptr[isub2*TILE + threadIdx.x] = (col < nl && f_row2 < nj) ? F[f_row2 * nl + col] : 0;
        } else {
            int isub1 = 2 * (threadIdx.y - TILE/2);
            int isub2 = isub1 + 1;
            for (int k = 0; k < TILE; k++) {
                acc1 += shEptr2[isub1*TILE + k] * shFptr2[k*TILE + threadIdx.x];
                acc2 += shEptr2[isub2*TILE + k] * shFptr2[k*TILE + threadIdx.x];
            }
        }

        DATA_TYPE* tmp;

        tmp = shEptr; shEptr = shEptr2; shEptr2 = tmp;
        tmp = shFptr; shFptr = shFptr2; shFptr2 = tmp;

        __syncthreads();
    }

    if (threadIdx.y >= TILE/2) {
        int isub1 = 2 * (threadIdx.y - TILE/2);
        int isub2 = isub1 + 1;
        for (int k = 0; k < TILE; k++) {
            acc1 += shEptr[isub1*TILE + k] * shFptr[k*TILE + threadIdx.x];
            acc2 += shEptr[isub2*TILE + k] * shFptr[k*TILE + threadIdx.x];
        }

        int row1 = blockIdx.y * blockDim.y + isub1;
        int row2 = blockIdx.y * blockDim.y + isub2;
        if (row1 < ni && col < nl) G[row1 * nl + col] = acc1;
        if (row2 < ni && col < nl) G[row2 * nl + col] = acc2;
    }
}

/* Main computational kernel on CPU */
void mm3_cpu(int ni, int nj, int nk, int nl, int nm,
        DATA_TYPE POLYBENCH_2D(E,NI,NJ,ni,nj),
        DATA_TYPE POLYBENCH_2D(A,NI,NK,ni,nk),
        DATA_TYPE POLYBENCH_2D(B,NK,NJ,nk,nj),
        DATA_TYPE POLYBENCH_2D(F,NJ,NL,nj,nl),
        DATA_TYPE POLYBENCH_2D(C,NJ,NM,nj,nm),
        DATA_TYPE POLYBENCH_2D(D,NM,NL,nm,nl),
        DATA_TYPE POLYBENCH_2D(G,NI,NL,ni,nl))
{
    int i, j, k;

    /* E := A*B */
    for (i = 0; i < _PB_NI; i++)
    {
        for (j = 0; j < _PB_NJ; j++)
        {
            E[i][j] = 0;
            for (k = 0; k < _PB_NK; ++k)
            {
                E[i][j] += A[i][k] * B[k][j];
            }
        }
    }

    /* F := C*D */
    for (i = 0; i < _PB_NJ; i++)
    {
        for (j = 0; j < _PB_NL; j++)
        {
            F[i][j] = 0;
            for (k = 0; k < _PB_NM; ++k)
            {
                F[i][j] += C[i][k] * D[k][j];
            }
        }
    }

    /* G := E*F */
    for (i = 0; i < _PB_NI; i++)
    {
        for (j = 0; j < _PB_NL; j++)
        {
            G[i][j] = 0;
            for (k = 0; k < _PB_NJ; ++k)
            {
                G[i][j] += E[i][k] * F[k][j];
            }
        }
    }
}


void mm3Cuda(int ni, int nj, int nk, int nl, int nm,
        DATA_TYPE POLYBENCH_2D(E,NI,NJ,ni,nj),
        DATA_TYPE POLYBENCH_2D(A,NI,NK,ni,nk),
        DATA_TYPE POLYBENCH_2D(B,NK,NJ,nk,nj),
        DATA_TYPE POLYBENCH_2D(F,NJ,NL,nj,nl),
        DATA_TYPE POLYBENCH_2D(C,NJ,NM,nj,nm),
        DATA_TYPE POLYBENCH_2D(D,NM,NL,nm,nl),
        DATA_TYPE POLYBENCH_2D(G,NI,NL,ni,nl),
        DATA_TYPE POLYBENCH_2D(G_outputFromGpu,NI,NL,ni,nl))
{
    DATA_TYPE *A_gpu;
    DATA_TYPE *B_gpu;
    DATA_TYPE *C_gpu;
    DATA_TYPE *D_gpu;
    DATA_TYPE *E_gpu;
    DATA_TYPE *F_gpu;
    DATA_TYPE *G_gpu;
    
    hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NI * NK);
    hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * NK * NJ);
    hipMalloc((void **)&C_gpu, sizeof(DATA_TYPE) * NJ * NM);
    hipMalloc((void **)&D_gpu, sizeof(DATA_TYPE) * NM * NL);
    hipMalloc((void **)&E_gpu, sizeof(DATA_TYPE) * NI * NJ);
    hipMalloc((void **)&F_gpu, sizeof(DATA_TYPE) * NJ * NL);
    hipMalloc((void **)&G_gpu, sizeof(DATA_TYPE) * NI * NL);

    hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NI * NK, hipMemcpyHostToDevice);
    hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * NK * NJ, hipMemcpyHostToDevice);
    hipMemcpy(C_gpu, C, sizeof(DATA_TYPE) * NJ * NM, hipMemcpyHostToDevice);
    hipMemcpy(D_gpu, D, sizeof(DATA_TYPE) * NM * NL, hipMemcpyHostToDevice);
    hipMemcpy(E_gpu, E, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyHostToDevice);
    hipMemcpy(F_gpu, F, sizeof(DATA_TYPE) * NJ * NL, hipMemcpyHostToDevice);
    hipMemcpy(G_gpu, G, sizeof(DATA_TYPE) * NI * NL, hipMemcpyHostToDevice);	
    
    // dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
    // dim3 grid1((size_t)(ceil( ((float)NJ) / ((float)DIM_THREAD_BLOCK_X) )),(size_t)(ceil((float)NI/ ((float)DIM_THREAD_BLOCK_Y) )));
    // dim3 grid2((size_t)(ceil( ((float)NL) / ((float)DIM_THREAD_BLOCK_X) )),(size_t)(ceil((float)NJ/ ((float)DIM_THREAD_BLOCK_Y) )));
    // dim3 grid3((size_t)(ceil( ((float)NL) / ((float)DIM_THREAD_BLOCK_X) )),(size_t)(ceil((float)NI/ ((float)DIM_THREAD_BLOCK_Y) )));

    // /* Start timer. */
    // polybench_start_instruments;

    // mm3_kernel1<<<grid1,block>>>(ni, nj, nk, nl, nm, A_gpu, B_gpu, E_gpu);
    // hipDeviceSynchronize();
    // mm3_kernel2<<<grid2,block>>>(ni, nj, nk, nl, nm, C_gpu, D_gpu, F_gpu);
    // hipDeviceSynchronize();
    // mm3_kernel3<<<grid3,block>>>(ni, nj, nk, nl, nm, E_gpu, F_gpu, G_gpu);
    // hipDeviceSynchronize();

    dim3 block(TILE, TILE);
    dim3 g1((NJ+TILE-1)/TILE, (NI+TILE-1)/TILE);
    dim3 g2((NL+TILE-1)/TILE, (NJ+TILE-1)/TILE);
    dim3 g3((NL+TILE-1)/TILE, (NI+TILE-1)/TILE);

    polybench_start_instruments;
    mm3_kernel1_tiled_pipelined<<<g1, block>>>(ni,nj,nk,A_gpu,B_gpu,E_gpu);
    hipDeviceSynchronize();
    mm3_kernel2_tiled_pipelined<<<g2, block>>>(nj,nl,nm,C_gpu,D_gpu,F_gpu);
    hipDeviceSynchronize();
    mm3_kernel3_tiled_pipelined<<<g3, block>>>(ni,nj,nl,E_gpu,F_gpu,G_gpu);
    hipDeviceSynchronize();
    polybench_stop_instruments;

    /* Stop and print timer. */
    printf("GPU Time in seconds:\n");
    polybench_print_instruments;
    hipMemcpy(G_outputFromGpu, G_gpu, sizeof(DATA_TYPE) * NI * NL, hipMemcpyDeviceToHost);
    
    hipFree(A_gpu);
    hipFree(B_gpu);
    hipFree(C_gpu);
    hipFree(D_gpu);
    hipFree(E_gpu);
    hipFree(F_gpu);
    hipFree(G_gpu);
}


/* DCE code. Must scan the entire live-out data.
Can be used also to check the correctness of the output. */
static
void print_array(int ni, int nl,
        DATA_TYPE POLYBENCH_2D(G,NI,NL,ni,nl))
{
int i, j;

for (i = 0; i < ni; i++)
    for (j = 0; j < nl; j++) {
    fprintf (stderr, DATA_PRINTF_MODIFIER, G[i][j]);
    if ((i * ni + j) % 20 == 0) fprintf (stderr, "\n");
    }
fprintf (stderr, "\n");
}


int main(int argc, char** argv)
{
    int ni = NI;
    int nj = NJ;
    int nk = NK;
    int nl = NL;
    int nm = NM;

    /* Variable declaration/allocation. */
    POLYBENCH_2D_ARRAY_DECL(E, DATA_TYPE, NI, NJ, ni, nj);
    POLYBENCH_2D_ARRAY_DECL(A, DATA_TYPE, NI, NK, ni, nk);
    POLYBENCH_2D_ARRAY_DECL(B, DATA_TYPE, NK, NJ, nk, nj);
    POLYBENCH_2D_ARRAY_DECL(F, DATA_TYPE, NJ, NL, nj, nl);
    POLYBENCH_2D_ARRAY_DECL(C, DATA_TYPE, NJ, NM, nj, nm);
    POLYBENCH_2D_ARRAY_DECL(D, DATA_TYPE, NM, NL, nm, nl);
    POLYBENCH_2D_ARRAY_DECL(G, DATA_TYPE, NI, NL, ni, nl);
    POLYBENCH_2D_ARRAY_DECL(G_outputFromGpu, DATA_TYPE, NI, NL, ni, nl);

    init_array(ni, nj, nk, nl, nm, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(D));

    GPU_argv_init();

    mm3Cuda(ni, nj, nk, nl, nm, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(D), POLYBENCH_ARRAY(E), 
        POLYBENCH_ARRAY(F), POLYBENCH_ARRAY(G), POLYBENCH_ARRAY(G_outputFromGpu));

    #ifdef RUN_ON_CPU

        /* Start timer. */
        polybench_start_instruments;

        mm3_cpu(ni, nj, nk, nl, nm, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(D), POLYBENCH_ARRAY(E), 
            POLYBENCH_ARRAY(F), POLYBENCH_ARRAY(G));
    
        /* Stop and print timer. */
        printf("CPU Time in seconds:\n");
        polybench_stop_instruments;
        polybench_print_instruments;

        compareResults(ni, nl, POLYBENCH_ARRAY(G), POLYBENCH_ARRAY(G_outputFromGpu));

    #else //print output to stderr so no dead code elimination

        print_array(ni, nl, POLYBENCH_ARRAY(G_outputFromGpu));

    #endif //RUN_ON_CPU


    POLYBENCH_FREE_ARRAY(A);
    POLYBENCH_FREE_ARRAY(B);
    POLYBENCH_FREE_ARRAY(C);
    POLYBENCH_FREE_ARRAY(D);
    POLYBENCH_FREE_ARRAY(E);
    POLYBENCH_FREE_ARRAY(F);
    POLYBENCH_FREE_ARRAY(G);
    POLYBENCH_FREE_ARRAY(G_outputFromGpu);

    return 0;
}

#include "../../common/polybench.c"


